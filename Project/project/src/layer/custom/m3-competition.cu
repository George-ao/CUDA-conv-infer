#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include <mma.h>
using namespace nvcuda;

#define TILE_WIDTH 16
#define BLOCK_SIZE 256
#define WARP_SIZE 32

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

#define L1_WMMA_M 8
#define L1_WMMA_N 32
#define L1_WMMA_K 16

#define L1_TILE_WIDTH 32
#define L1_TILE_HEIGHT 8

__global__ void layer1_matrix_mul_built_in_unrolling_kernel(float * __restrict__ device_output, const float * __restrict__ 
    device_input, const float * __restrict__ device_mask, 
    const int Batch, const int Map_out, const int Channel, 
    const int Height, const int Width, const int K)
{
    __shared__ half tileA[L1_WMMA_M][L1_WMMA_K];
    __shared__ half tileB[L1_WMMA_K][L1_WMMA_N];
    __shared__ float tileC[L1_WMMA_M][L1_WMMA_N];
    wmma::fragment<wmma::matrix_a, L1_WMMA_M, L1_WMMA_N, L1_WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, L1_WMMA_M, L1_WMMA_N, L1_WMMA_K, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, L1_WMMA_M, L1_WMMA_N, L1_WMMA_K, float> c_frag;
    wmma::fill_fragment(c_frag, 0.0f);
    // load and unroll input 
    int by = blockIdx.y;
    int bx = blockIdx.x;
    int ty = threadIdx.y;
    int tx = threadIdx.x;
    int cur_batch = blockIdx.z; 

    int row = by * L1_TILE_HEIGHT + ty;
    int col = bx * L1_TILE_WIDTH + tx;

    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;

    int h_w = Height * Width;
    int c_h_w = Channel * h_w;
    int hout_wout = Height_out * Width_out;
    int m_hout_wout = Map_out * hout_wout;

    int cb_chw = cur_batch * c_h_w;
    // #define in_4d(i3, i2, i1, i0) device_input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    // #define out_3d(i3, i2, i1) device_output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + i1]
    #define out_3d(i3, i2, i1) device_output[(i3) * (m_hout_wout) + (i2) * (hout_wout) + i1]

    // perform multiplication
    int numARows = Map_out;
    int numAColumns = Channel * K * K;
    int numBRows = numAColumns;
    int numBColumns =  hout_wout; // Height_out * Width_out;
    int numCRows = numARows;
    int numCColumns = numBColumns;

    // reuse.1: reuse & remove size_t for 5000
    int K_square = K * K;
    int h = col / Width_out;
    int w = col - h * Width_out;

    #pragma unroll
    for (int tileId = 0; tileId < (numAColumns - 1) / L1_WMMA_K + 1; tileId++)
    {
        if (tx < 16)
        {
            if (row < numARows && tileId * L1_WMMA_K + tx < numAColumns) tileA[ty][tx] = device_mask[row * numAColumns + tileId * L1_WMMA_K + tx];
            else tileA[ty][tx] = 0;
        }

        // two steps to load = L1_WMMA_K / TILE_HEIGHT
        #pragma unroll
        for (int i=0; i< 2; i++)
        {
            if (col < numBColumns && (tileId * L1_WMMA_K) + ty + i * L1_TILE_HEIGHT < numBRows) 
            {
                int cur_row = (tileId * L1_WMMA_K) + ty + i * L1_TILE_HEIGHT;
                int c = cur_row / K_square;
                int offset = cur_row % K_square;
                int p = offset / K;
                int q = offset - p * K;
                // tileB[ty][tx] = in_4d(cur_batch, c, h + p, w + q);
                tileB[ty + i * L1_TILE_HEIGHT][tx] = device_input[cb_chw + c * h_w + (h + p) * Width + w + q];
            } 
            else tileB[ty + i * L1_TILE_HEIGHT][tx] = 0;
        }
        __syncthreads();
        
        if (ty < 1)
        {
            wmma::load_matrix_sync(a_frag, &tileA[0][0], L1_WMMA_K);
            wmma::load_matrix_sync(b_frag, &tileB[0][0], L1_WMMA_N);
            wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
        }
        __syncthreads();
    }

    if (ty < 1) wmma::store_matrix_sync(&tileC[0][0], c_frag, L1_WMMA_N, wmma::mem_row_major);
    __syncthreads();
    if (row < numCRows && col < numCColumns) out_3d(cur_batch, row, col) = tileC[ty][tx];
    // #undef in_4d
    #undef out_3d
}


__global__ void matrix_mul_built_in_unrolling_kernel(float * __restrict__ device_output, const float * __restrict__ 
    device_input, const float * __restrict__ device_mask, 
    const int Batch, const int Map_out, const int Channel, 
    const int Height, const int Width, const int K)
{
    __shared__ half tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ half tileB[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileC[TILE_WIDTH][TILE_WIDTH];
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
    wmma::fill_fragment(c_frag, 0.0f);
    // load and unroll input 
    int by = blockIdx.y;
    int bx = blockIdx.x;
    int ty = threadIdx.y;
    int tx = threadIdx.x;
    int cur_batch = blockIdx.z; 

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;

    int h_w = Height * Width;
    int c_h_w = Channel * h_w;
    int hout_wout = Height_out * Width_out;
    int m_hout_wout = Map_out * hout_wout;

    int cb_chw = cur_batch * c_h_w;
    // #define in_4d(i3, i2, i1, i0) device_input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    // #define out_3d(i3, i2, i1) device_output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + i1]
    #define out_3d(i3, i2, i1) device_output[(i3) * (m_hout_wout) + (i2) * (hout_wout) + i1]

    // perform multiplication
    // const float *A = device_mask;
    
    int numARows = Map_out;
    int numAColumns = Channel * K * K;
    int numBRows = numAColumns;
    int numBColumns =  hout_wout; // Height_out * Width_out;
    int numCRows = numARows;
    int numCColumns = numBColumns;

    // reuse.1: reuse & remove size_t for 5000
    int K_square = K * K;
    int h = col / Width_out;
    int w = col - h * Width_out;

    #pragma unroll
    for (int tileId = 0; tileId < (numAColumns - 1) / TILE_WIDTH + 1; tileId++) 
    {
        if (row < numARows && tileId * TILE_WIDTH + tx < numAColumns) tileA[ty][tx] = device_mask[ row * numAColumns + tileId * TILE_WIDTH + tx];
        else tileA[ty][tx] = 0;

        if (col < numBColumns && tileId * TILE_WIDTH + ty < numBRows) 
        {

            int cur_row = tileId * TILE_WIDTH + ty;
            int c = cur_row / K_square;
            int offset = cur_row % K_square;
            int p = offset / K;
            int q = offset - p * K;

            // tileB[ty][tx] = in_4d(cur_batch, c, h + p, w + q);
            tileB[ty][tx] = device_input[cb_chw + c * h_w + (h + p) * Width + w + q];
        } 
        else tileB[ty][tx] = 0;

        __syncthreads();
        
        if (ty < 2)
        {
            wmma::load_matrix_sync(a_frag, &tileA[0][0], TILE_WIDTH);
            wmma::load_matrix_sync(b_frag, &tileB[0][0], TILE_WIDTH);
            wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
        }

        __syncthreads();
    }

    if (ty < 2) wmma::store_matrix_sync(&tileC[0][0], c_frag, TILE_WIDTH, wmma::mem_row_major);
    __syncthreads();
    if (row < numCRows && col < numCColumns) out_3d(cur_batch, row, col) = tileC[ty][tx];

    // #undef in_4d
    #undef out_3d
}


__host__ void GPUInterface::conv_forward_gpu_prolog(const float * __restrict__ host_output, const float * __restrict__ 
    host_input, const float * __restrict__ host_mask, float ** __restrict__ device_output_ptr, float ** __restrict__ 
    device_input_ptr, float ** __restrict__ device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;

    int input_size =  (Batch * Channel * Height * Width) * sizeof(float);
    int output_size = (Batch * Map_out * Height_out * Width_out) * sizeof(float);
    int mask_size = (Map_out * Channel * K * K) * sizeof(float);

    hipMalloc((void **)device_input_ptr, input_size);
    hipMalloc((void **)device_output_ptr, output_size);
    hipMalloc((void **)device_mask_ptr, mask_size);

    hipMemcpy(*device_input_ptr, host_input, input_size, hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, mask_size, hipMemcpyHostToDevice);
}


__host__ void GPUInterface::conv_forward_gpu(float * __restrict__ device_output, const float * __restrict__ 
    device_input, const float * __restrict__ device_mask, const int Batch, 
    const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    int W_grid, H_grid;

    // lay1
    if (Map_out == 4)
    {
        W_grid = ceil(1.0 * Height_out * Width_out / L1_WMMA_N);
        H_grid = ceil(1.0 * Map_out / L1_WMMA_M);
        dim3 L1_Dimblock(L1_TILE_WIDTH, L1_TILE_HEIGHT, 1);
        dim3 L1_Dimgrid(W_grid, H_grid, Batch);
        layer1_matrix_mul_built_in_unrolling_kernel<<<L1_Dimgrid, L1_Dimblock>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);
    }
    // lay2
    else
    {
        W_grid = ceil(1.0 * Height_out * Width_out / TILE_WIDTH);
        H_grid = ceil(1.0 * Map_out / TILE_WIDTH);
        dim3 Dimblock(TILE_WIDTH, TILE_WIDTH, 1);
        // dim3 Dimblock(16, 2, 1);
        dim3 Dimgrid(W_grid, H_grid, Batch);
        matrix_mul_built_in_unrolling_kernel<<<Dimgrid, Dimblock>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);
    }
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float * __restrict__ host_output, float * __restrict__ device_output, 
    float * __restrict__ device_input, float * __restrict__ device_mask, const int Batch, const int Map_out, 
    const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Copy the output back to host
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    int output_size = (Batch * Map_out * Height_out * Width_out) * sizeof(float);
    hipMemcpy(host_output, device_output, output_size, hipMemcpyDeviceToHost);

    // TODO: Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}